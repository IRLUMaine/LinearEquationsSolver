#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define RTHRESHOLD .0001
#define THRESHOLD .0001


#define DISP -1

typedef float MatrixType;
__global__ void computeIter(int maxRow, int num, int size, int iter, MatrixType *dXs, MatrixType *dNextXs, int *dIndexs, MatrixType *dRowVals, int *dRowInds, MatrixType *dYs, int *dDiffs);


void computeIterations(int varPBlock, int maxRow, int num, int size, int iter, MatrixType *dXs, MatrixType *dNextXs, int *dIndexs, MatrixType *dRowVals, int *dRowInds, MatrixType *dYs, int* dDiffs) {
	int blocks = (num / (varPBlock));
    hipError_t error;


	if ((blocks * (varPBlock)) != num) blocks++;
	if (varPBlock != 1024) printf("INCORRECT PARAMETER varPBlock VALID VALUES=1024\n");

	computeIter<<<blocks, 1024, 12288>>>(maxRow, num, size, iter, dXs, dNextXs, dIndexs, dRowVals, dRowInds, dYs, dDiffs);
    if ((error = hipDeviceSynchronize()) != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(error));
		exit(1);
        return;
    }
}

#define NWEIGHT 1
#define LWEIGHT 4

__global__ void computeIter(int maxRow, int num, int size, int iter, MatrixType *dXs, MatrixType *dNextXs, int *dIndexs, MatrixType *dRowVals, int *dRowInds, MatrixType *dYs, int *dDiffs) {
	__shared__ int shared[3072];
	// Each iteration x val for this block
	MatrixType *x = (MatrixType*)&shared[0];
	int *inds = &shared[1024];

	// diffs for last iteration
	int *diffs = (int*)&shared[2048];

	// index in the x array for the changing vars
	int localIndex[20];
	MatrixType localCoeff[20];

	// Local x value
	MatrixType lx;

	// variables that are not changing
	// first is the A[i][i] diag val
	// second is a value that will be precalculated
	// for the vars that aren't changing
	MatrixType staticVar0;
	MatrixType staticVar1;

	// global index of this thread
	int loc = blockDim.x * blockIdx.x + threadIdx.x;
	if (loc >= num) {
		return;
	}
	// local index of this thread
	int id = threadIdx.x;
	// row to operate on
	int ind = dIndexs[loc];
	//printf("loc:%d id:%d ind:%d %p\n", loc, id, ind, dIndexs);

	inds[id] = ind;
	// addresses for sparse row data
	MatrixType* dRowV = dRowVals + (ind * maxRow);
	int* dRowI = dRowInds + (ind * maxRow);

	// counting
	int i, j;
	// flaggging
	int flag;
	// temp stuff
	int temp;
	MatrixType tempF;
	// localIndex counting
	int ct = 0;
	lx = x[id] = dXs[loc];
	__syncthreads();
	diffs[0] = 0;

	// setup statics
	for (i = 0; dRowI[i] != -1; i++) {
		if (dRowI[i] == ind) {
			staticVar0 = dRowV[i];
/*
			if (loc == DISP) {
				printf("Set A[i][i] to %lf\n", staticVar0);
			}
*/
		}
	}
	staticVar1 = dYs[ind];// / staticVar0;
/*
	if (loc == DISP) {
		printf("Set Y to %lf\n", staticVar1);
	}
*/
/*
	if (loc == 5222) {
		for (i = 0; dRowI[i] != -1; i++) {
			printf("%lf * x_%d + ", dRowV[i], dRowI[i]);
		}
		printf("\n\n");
	}
*/

	// search for statics / dynamics
	for (j = 0; (j < maxRow) && (dRowI[j] != -1); j++) {
		temp = dRowI[j];
		if (temp != ind) {
			flag = 1;
			for (i = 0; i < 1024; i++) {
				if (temp == inds[i]) {
					flag = 0;
					localCoeff[ct] = dRowV[j];
					localIndex[ct++] = i;
/*
					if (loc == DISP) {
						printf("Dyn -= %lf * x_%d\n", localCoeff[ct-1], localIndex[ct-1]);
					}
*/
					break;
				}
			}
			if (flag) {
				staticVar1 -= dRowV[j] * dXs[temp];// / staticVar0;
/*
				if (loc == DISP) {
					printf("-= %lf * %lf = %lf\n", dRowV[j], dXs[temp], staticVar1);
				}
*/
			}
		}
	}
	localIndex[ct] = -1;


	// last iteration need to store the diffs for convergence
	// checking
	tempF = staticVar1;
	for (j = 0; localIndex[j] != -1; j++) {
		tempF -= localCoeff[j] * x[localIndex[j]];
	}
	//tempF += staticVar1;
	tempF /= staticVar0;
	tempF = (NWEIGHT * tempF + LWEIGHT * lx) / (NWEIGHT + LWEIGHT);
	//diffs[0] = tempF;
	if (fabs(lx - tempF) > (RTHRESHOLD * fabs(tempF) + THRESHOLD)) {

		diffs[0] = 1;//fabs(tempF - x[id]) / fabs(x[id]);
	//} else {
	//	diffs[id] = 0;
	}
	lx = x[id] = tempF;
	__syncthreads();

	// time for real calculations... damn
	// yes this loop runs iter - 1 times
	// last time requires diff calcs
	for (i = 1; i < iter; i++) {
		tempF = staticVar1;
		for (j = 0; localIndex[j] != -1; j++) {
			tempF -= localCoeff[j] * x[localIndex[j]];
		}
		//tempF += staticVar1;
		tempF /= staticVar0;
		tempF = (NWEIGHT * tempF + LWEIGHT * lx) / (NWEIGHT + LWEIGHT);
		lx = x[id] = tempF;
		__syncthreads();
	}
/*
	tempF = dYs[ind];
	if (loc == DISP) {
		printf("Val: %lf\n", tempF);
	}
	for (j = 0; dRowI[j] != -1; j++) {
		if (dRowI[j] != ind) {
			tempF -= dRowV[j] * dXs[dRowI[j]];
		} else {
			staticVar1 = dRowV[j];
		}
	if (loc == DISP) {
		printf("Val: %lf\n", tempF);
	}
	}
	tempF /= staticVar0;
	if (loc == DISP) {
		printf("Val: %lf %lf\n", tempF, dXs[ind]);
	}
	//tempF += staticVar1;
	tempF = NWEIGHT * tempF / (NWEIGHT + LWEIGHT) + LWEIGHT * dXs[ind] / (NWEIGHT + LWEIGHT);
	//diffs[0] = tempF;
	diffs[id] = fabs(tempF - dXs[ind]);/// / x[id];
	x[id] = tempF;
	__syncthreads();
*/

	// diff to max reduction
/*
	for (i = 1; i < 1024; i <<= 1) {
		if ((id & i) == 0) {
			if (diffs[id] < diffs[id + i]) {
				diffs[id] = diffs[id + i];
			}
		} else {
			break;
		}
		__syncthreads();
	}
*/
//	if (diffs[id] > 0) {
//		diffs[0] = 1;
//	}
/*
	__syncthreads();
	if (loc == DISP) {
		printf("Was: %lf Now: %lf\n", dXs[loc], x[id]);
	}
*/
	dXs[loc] = x[id];
	if (id == 0) {
		dDiffs[blockIdx.x] = diffs[0];
	}
}
