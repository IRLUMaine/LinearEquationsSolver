#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define RTHRESHOLD .00001
#define THRESHOLD .00001

//#define FINDEX

#define DISP 50001
//126976
//167940

//#define single(x) if (loc == DISP) x
#define single(x)

typedef float MatrixType;
__global__ void computeIter(int id, int maxRow, int num, int iter,
		MatrixType *dXs, MatrixType *dNextXs, int *dIndexs,
		MatrixType *dRowVals, int *dRowInds, MatrixType *dYs, int *dDiffs);

#ifdef FINDEX
__global__ void rearrange(int id, int maxRow, int num, int iter,
		MatrixType *dXs, MatrixType *dNextXs, int *dIndexs,
		MatrixType *dRowVals, MatrixType* dRowVRearr, int *dRowInds,
		int *dRowIRearr, MatrixType *dYs, int *dDiffs);
#endif

void rearrangeCall(int id, int varPBlock, int maxRow, int num, int size,
		int iter, MatrixType *dXs, MatrixType *dNextXs, int *dIndexs,
		MatrixType *dRowVals, MatrixType* dRowVRearr, int *dRowInds,
		int *dRowIRearr, MatrixType *dYs, int* dDiffs) {
#ifdef FINDEX
	int blocks = (num / (varPBlock));
	hipError_t error;

	if ((blocks * (varPBlock)) != num)
		blocks++;
	if (varPBlock != 1024)
		printf("INCORRECT PARAMETER varPBlock VALID VALUES=1024\n");

	printf("Rearranging...");
	rearrange<<<blocks, 1024, 12288>>>(id, maxRow, num, iter, dXs, dNextXs,
			dIndexs, dRowVals, dRowVRearr, dRowInds, dRowIRearr, dYs, dDiffs);
	if ((error = hipDeviceSynchronize()) != hipSuccess) {
		printf("Rearrange Error %d: %s\n", id, hipGetErrorString(error));
		exit(1);
		return;
	}
	printf("Done\n");
#endif
}

void computeIterations(int id, int varPBlock, int maxRow, int num, int size,
		int iter, MatrixType *dXs, MatrixType *dNextXs, int *dIndexs,
		MatrixType *dRowVals, MatrixType* dRowVRearr, int *dRowInds,
		int *dRowIRearr, MatrixType *dYs, int* dDiffs) {
	int blocks = (num / (varPBlock));
	hipError_t error;

	if ((blocks * (varPBlock)) != num)
		blocks++;
	if (varPBlock != 1024)
		printf("INCORRECT PARAMETER varPBlock VALID VALUES=1024\n");

#ifdef FINDEX
	computeIter<<<blocks, 1024, 12288>>>(id, maxRow, num, iter, dXs, dNextXs,
			dIndexs, dRowVRearr, dRowIRearr, dYs, dDiffs);
#else
	computeIter<<<blocks, 1024, 12288>>>(id, maxRow, num, iter, dXs, dNextXs,
			dIndexs, dRowVals, dRowInds, dYs, dDiffs);
#endif

	if ((error = hipDeviceSynchronize()) != hipSuccess) {
		printf("Error %d: %s\n", id, hipGetErrorString(error));
		exit(1);
		return;
//    } else {
//    	printf("Iter\n");
	}
}

#ifdef FINDEX
__global__ void rearrange(int gid, int maxRow, int num, int iter,
		MatrixType *dXs, MatrixType *dNextXs, int *dIndexs,
		MatrixType *dRowVals, MatrixType* dRowVRearr, int *dRowInds,
		int *dRowIRearr, MatrixType *dYs, int *dDiffs) {
	__shared__ int shared[3072];
	int id = threadIdx.x;
	int loc = blockDim.x * blockIdx.x + threadIdx.x;
	if (loc >= num) {
		return;
	}
	// Each iteration x val for this block
	int *inds = &shared[1024];

	// diffs for last iteration
	int *diffs = (int*) &shared[2048];

	// index in the x array for the changing vars
	int localIndex[20];
	MatrixType localCoeff[20];

	// index in the x array for the static vars
	int staticIndex[20];
	MatrixType staticCoeff[20];
	MatrixType staticVar0;
	//MatrixType staticVar1;

	// global index of this thread
	// local index of this thread
	// row to operate on
	int ind = dIndexs[loc];
	//printf("loc:%d id:%d ind:%d %p\n", loc, id, ind, dIndexs);

	inds[id] = ind;
	// addresses for sparse row data
	MatrixType* dRowV = dRowVals + (loc * maxRow);
	int* dRowI = dRowInds + (loc * maxRow);

	// counting
	int i, j;
	// flaggging
	int flag;
	// temp stuff
	int temp;
	// localIndex counting
	int ct = 0;
	// staticIndex counting
	int sct = 0;
	__syncthreads();
	diffs[0] = 0;

	// setup statics
	for (i = 0; dRowI[i] != -1; i++) {
		if (dRowI[i] == ind) {
			staticVar0 = dRowV[i];
		}
	}
	__syncthreads();
	// search for statics / dynamics
	for (j = 0; (j < maxRow) && (dRowI[j] != -1); j++) {
		temp = dRowI[j];
		if (temp != ind) {
			flag = 1;
			for (i = 0; i < 1024; i++) {
				if (temp == inds[i]) {
					flag = 0;
					localCoeff[ct] = dRowV[j];
					localIndex[ct++] = i;
					break;
				}
			}
			if (flag) {
				staticCoeff[sct] = dRowV[j];
				staticIndex[sct++] = -temp - 1;
			}
		}
	}
	localIndex[ct] = -1;
	staticIndex[sct] = 1;
	__syncthreads();
	dRowIRearr[loc] = ct + sct;
	dRowVRearr[loc] = staticVar0;
	sct = loc + num;
	for (ct = 0; localIndex[ct] != -1; ct++) {
		dRowIRearr[sct] = localIndex[ct];
		dRowVRearr[sct] = localCoeff[ct];
		sct += num;
	}
	__syncthreads();
	for (ct = 0; staticIndex[ct] != 1; ct++) {
		dRowIRearr[sct] = staticIndex[ct];
		dRowVRearr[sct] = staticCoeff[ct];
		sct += num;
	}
}
#endif

#define NWEIGHT 1
#define LWEIGHT 4

__global__ void computeIter(int gid, int maxRow, int num, int iter,
		MatrixType *dXs, MatrixType *dNextXs, int *dIndexs,
		MatrixType *dRowVals, int *dRowInds, MatrixType *dYs, int *dDiffs) {
	__shared__ int shared[3072];
	int loc = blockDim.x * blockIdx.x + threadIdx.x;
	// local index of this thread
	int id = threadIdx.x;
	if (loc < num) {
		// Each iteration x val for this block
		MatrixType *x = (MatrixType*) &shared[0];
		int *inds = &shared[1024];

		// diffs for last iteration
		int *diffs = (int*) &shared[2048];

		// index in the x array for the changing vars
		int localIndex[20];
		MatrixType localCoeff[20];

		// Local x value
		MatrixType lx;

		// variables that are not changing
		// first is the A[i][i] diag val
		// second is a value that will be precalculated
		// for the vars that aren't changing
		MatrixType staticVar0;
		MatrixType staticVar1;

		// global index of this thread
		// row to operate on
		int ind = dIndexs[loc];
		//printf("loc:%d id:%d ind:%d %p\n", loc, id, ind, dIndexs);

		inds[id] = ind;

		// counting
		int i, j;
		MatrixType tempF;
		// localIndex counting
		int ct = 0;
		lx = x[id] = dXs[ind];
		__syncthreads();
		diffs[0] = 0;

		staticVar1 = dYs[loc];
		__syncthreads();

#ifdef FINDEX
		int nvars = dRowInds[loc];
		int dInd = loc + num;
		staticVar0 = dRowVals[loc];
		for (i = 0; i < nvars; i++) {

			int curIndex = dRowInds[dInd];
			MatrixType curValue = dRowVals[dInd];

			if (curIndex >= 0) {
				localCoeff[ct] = curValue;
				localIndex[ct++] = curIndex;
			} else {
				staticVar1 -= curValue * dXs[-curIndex - 1];
				staticVar1 -= 1;
			}
			dInd += num;
		}
#else
		// addresses for sparse row data
		MatrixType* dRowV = dRowVals + (loc * maxRow);
		int* dRowI = dRowInds + (loc * maxRow);
		// temp stuff
		int temp;
		// flaggging
		int flag;

		// search for statics / dynamics
		for (j = 0; (j < maxRow) && (dRowI[j] != -1); ++j) {
			temp = dRowI[j];
			if (temp != ind) {
				flag = 1;
				for (i = 0; i < 1024; ++i) {
					if (temp == inds[i]) {
						flag = 0;
						localCoeff[ct] = dRowV[j];
						localIndex[ct++] = i;
						break;
					}
				}
				if (flag) {
					staticVar1 -= dRowV[j] * dXs[temp];
				}
			} else {
				staticVar0 = dRowV[j];
			}
		}
#endif
		localIndex[ct] = -1;

		// last iteration need to store the diffs for convergence
		// checking
		tempF = staticVar1;
		for (j = 0; localIndex[j] != -1; ++j) {
			tempF -= localCoeff[j] * x[localIndex[j]];
		}
		tempF /= staticVar0;
		tempF = (NWEIGHT * tempF + LWEIGHT * lx) / (NWEIGHT + LWEIGHT);
		if (fabs(lx - tempF) > (RTHRESHOLD * fabs(tempF) + THRESHOLD)) {
			diffs[0] = 1;
		}
		lx = x[id] = tempF;
		__syncthreads();

		// time for real calculations... damn
		// yes this loop runs iter - 1 times
		// first time requires diff calcs
		for (i = 1; i < iter; i++) {
			tempF = staticVar1;
			for (j = 0; localIndex[j] != -1; ++j) {
				tempF -= localCoeff[j] * x[localIndex[j]];
			}
			//tempF += staticVar1;
			tempF /= staticVar0;
			tempF = (NWEIGHT * tempF + LWEIGHT * lx) / (NWEIGHT + LWEIGHT);
			lx = x[id] = tempF;
			__syncthreads();
		}

		dXs[ind] = x[id];
		if (id == 0) {
			dDiffs[blockIdx.x] = diffs[0];
		}
	}
}
