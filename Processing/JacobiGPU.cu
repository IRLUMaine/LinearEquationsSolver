
#include <hip/hip_runtime.h>

typedef float MatrixType;
__global__ void computeIter(int maxRow, MatrixType *dXs, MatrixType *dNextXs, int *dIndexs, MatrixType *dRowVals, int *dRowInds, MatrixType *dYs);


void computeIterations(int maxRow, MatrixType *dXs, MatrixType *dNextXs, int *dIndexs, MatrixType *dRowVals, int *dRowInds, MatrixType *dYs) {
	computeIter<<<1, 1>>>(maxRow, dXs, dNextXs, dIndexs, dRowVals, dRowInds, dYs);
}

__global__ void computeIter(int maxRow, MatrixType *dXs, MatrixType *dNextXs, int *dIndexs, MatrixType *dRowVals, int *dRowInds, MatrixType *dYs) {

}
